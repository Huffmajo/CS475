#include "hip/hip_runtime.h"
/***********************************************************
 * Program: project6.cu
 * Author: Joel Huffman
 * Last updated: 5/25/2019
 * Sources: http://web.engr.oregonstate.edu/~mjb/cs575/Projects/proj06.html
 ***********************************************************/

// System includes
#include <stdio.h>
#include <assert.h>
#include <malloc.h>
#include <math.h>
#include <stdlib.h>
#include <time.h>

// CUDA runtime
#include <hip/hip_runtime.h>

// Helper functions and utilities to work with CUDA
#include "helper_functions.h"
#include "hip/hip_runtime_api.h"


#ifndef BLOCKSIZE
#define BLOCKSIZE		32		// number of threads per block
#endif

#ifndef SIZE
#define SIZE			1*1024*1024	// array size
#endif

#ifndef NUMTRIALS
#define NUMTRIALS		100		// to make the timing more accurate
#endif

#ifndef TOLERANCE
#define TOLERANCE		0.00001f	// tolerance to relative error
#endif

// ranges for the random numbers:
const float XCMIN =	 0.0;
const float XCMAX =	 2.0;
const float YCMIN =	 0.0;
const float YCMAX =	 2.0;
const float RMIN  =	 0.5;
const float RMAX  =	 2.0;

float Ranf( float low, float high )
{
        float r = (float) rand();               // 0 - RAND_MAX
        float t = r  /  (float) RAND_MAX;       // 0. - 1.

        return   low  +  t * ( high - low );
}

int Ranf( int ilow, int ihigh )
{
        float low = (float)ilow;
        float high = ceil( (float)ihigh );

        return (int) Ranf(low,high);
}

void TimeOfDaySeed( )
{
	struct tm y2k = { 0 };
	y2k.tm_hour = 0;   y2k.tm_min = 0; y2k.tm_sec = 0;
	y2k.tm_year = 100; y2k.tm_mon = 0; y2k.tm_mday = 1;

	time_t  timer;
	time( &timer );
	double seconds = difftime( timer, mktime(&y2k) );
	unsigned int seed = (unsigned int)( 1000.*seconds );    // milliseconds
	srand( seed );
}

// determine if vertical plate is hit by laser
__global__  void MonteCarlo( float *A, float *B, float *C, float *D )
{
/*
	__shared__ float numHits[BLOCKSIZE];
	unsigned int numItems = blockDim.x;
	unsigned int tnum = threadIdx.x;
	unsigned int wgNum = blockIdx.x;
*/
	unsigned int gid = blockIdx.x*blockDim.x + threadIdx.x;

	// randomize the location and radius of the circle:
	float xc = A[gid];
	float yc = B[gid];
	float r =  C[gid];

	// solve for the intersection using the quadratic formula:
	float a = 2.;
	float b = -2.*( xc + yc );
	float c = xc*xc + yc*yc - r*r;
	float d = b*b - 4.*a*c;

	// If d is less than 0, then the circle was completely missed (Case A) 
	if (d >= 0.)
	{
		// hits the circle:
		// get the first intersection:
		d = sqrtf( d );
		float t1 = (-b + d ) / ( 2.*a );	// time to intersect the circle
		float t2 = (-b - d ) / ( 2.*a );	// time to intersect the circle
		float tmin = t1 < t2 ? t1 : t2;		// only care about the first intersection

		// If tmin is less than 0., then the circle completely engulfs the laser pointer (Case B)
		if (tmin >= 0.)
		{
			// where does it intersect the circle?
			float xcir = tmin;
			float ycir = tmin;

			// get the unitized normal vector at the point of intersection:
			float nx = xcir - xc;
			float ny = ycir - yc;
			float n = sqrtf( nx*nx + ny*ny );
			nx /= n;	// unit vector
			ny /= n;	// unit vector

			// get the unitized incoming vector:
			float inx = xcir - 0.;
			float iny = ycir - 0.;
			float in = sqrtf( inx*inx + iny*iny );
			inx /= in;	// unit vector
			iny /= in;	// unit vector

			// get the outgoing (bounced) vector:
			float dot = inx*nx + iny*ny;
//			float outx = inx - 2.*nx*dot;	// angle of reflection = angle of incidence`
			float outy = iny - 2.*ny*dot;	// angle of reflection = angle of incidence`

			// find out if it hits the infinite plate:
			float t = ( 0. - ycir ) / outy;

			// If t is less than 0., then the reflected beam went up instead of down (Case C)
			if (t >= 0.)
			{
				D[gid] = 1;;
			}
		}
	}

/*
	prods[tnum] = A[gid] * B[gid];

	for (int offset = 1; offset < numItems; offset *= 2)
	{
		int mask = 2 * offset - 1;
		__syncthreads();
		if ((tnum & mask) == 0)
		{
			numHits[tnum] += numHits[tnum + offset];
		}
	}

	__syncthreads();
	if (tnum == 0)
		D[wgNum] = numHits[0];
*/
}


// main program:

int
main( int argc, char* argv[ ] )
{
//	int dev = findCudaDevice(argc, (const char **)argv);

	// allocate host memory:
	float *xcs = new float [ NUMTRIALS ];
	float *ycs = new float [ NUMTRIALS ];
	float *rs = new float [ NUMTRIALS ];
	float *hits = new float [ NUMTRIALS ];

	// fill arrays with random values in range
	for( int n = 0; n < NUMTRIALS; n++ )
	{
		xcs[n] = Ranf( XCMIN, XCMAX );
                ycs[n] = Ranf( YCMIN, YCMAX );
                rs[n] = Ranf(  RMIN,  RMAX ); 
		hits[n] = 0.;
	}

	// allocate device memory:

	float *dxcs, *dycs, *drs, *dhits;

	dim3 dimsxcs( NUMTRIALS, 1, 1 );
	dim3 dimsycs( NUMTRIALS, 1, 1 );
	dim3 dimsrc( NUMTRIALS, 1, 1 );
	dim3 dimshits( NUMTRIALS, 1, 1 );

	//__shared__ float prods[SIZE/BLOCKSIZE];


	hipError_t status;
	status = hipMalloc( reinterpret_cast<void **>(&dxcs), NUMTRIALS*sizeof(float) );
		checkCudaErrors( status );
	status = hipMalloc( reinterpret_cast<void **>(&dycs), NUMTRIALS*sizeof(float) );
		checkCudaErrors( status );
	status = hipMalloc( reinterpret_cast<void **>(&drs), NUMTRIALS*sizeof(float) );
		checkCudaErrors( status );
	status = hipMalloc( reinterpret_cast<void **>(&dhits), NUMTRIALS*sizeof(float) );
		checkCudaErrors( status );

	// copy host memory to the device:

	status = hipMemcpy( dxcs, xcs, NUMTRIALS*sizeof(float), hipMemcpyHostToDevice );
		checkCudaErrors( status );
	status = hipMemcpy( dycs, ycs, NUMTRIALS*sizeof(float), hipMemcpyHostToDevice );
		checkCudaErrors( status );
	status = hipMemcpy( drs, rs, NUMTRIALS*sizeof(float), hipMemcpyHostToDevice );
		checkCudaErrors( status );
	status = hipMemcpy( dhits, hits, NUMTRIALS*sizeof(float), hipMemcpyHostToDevice );
		checkCudaErrors( status );

	// setup the execution parameters:

	dim3 threads(BLOCKSIZE, 1, 1 );
	dim3 grid( NUMTRIALS / threads.x, 1, 1 );

	// Create and start timer

	hipDeviceSynchronize( );

	// allocate CUDA events that we'll use for timing:

	hipEvent_t start, stop;
	status = hipEventCreate( &start );
		checkCudaErrors( status );
	status = hipEventCreate( &stop );
		checkCudaErrors( status );

	// record the start event:

	status = hipEventRecord( start, NULL );
		checkCudaErrors( status );

	// execute the kernel:

//	for( int t = 0; t < NUMTRIALS; t++)
//	{
	        MonteCarlo<<< grid, threads >>>( dxcs, dycs, drs, dhits );
//	}

	// record the stop event:

	status = hipEventRecord( stop, NULL );
		checkCudaErrors( status );

	// wait for the stop event to complete:

	status = hipEventSynchronize( stop );
		checkCudaErrors( status );

	float msecTotal = 0.0f;
	status = hipEventElapsedTime( &msecTotal, start, stop );
		checkCudaErrors( status );

	// copy result from the device to the host:

	status = hipMemcpy( hits, dhits, NUMTRIALS*sizeof(float), hipMemcpyDeviceToHost );
		checkCudaErrors( status );

	// add up all the hits
	int numHits = 0;
	for (int i = 0; i < NUMTRIALS; i++)
	{
		if (hits[i] == 1)
		{
			numHits++;
		}
	}

	// compute and print the performance
	double secondsTotal = 0.001 * (double)msecTotal;
	double multsPerSecond = (float)NUMTRIALS / secondsTotal;
	double megaMultsPerSecond = multsPerSecond / 1000000.;
	double probability = (float)numHits / (float)NUMTRIALS;

	// print performance
	printf("%d\t%d\t%lf\t%lf\n", NUMTRIALS, BLOCKSIZE, megaMultsPerSecond, probability);

	// also write performance to results.txt
	FILE *fp;
	fp = fopen("results.txt", "a");
	fprintf(fp, "%d\t%d\t%lf\t%lf\n", NUMTRIALS, BLOCKSIZE, megaMultsPerSecond, probability);
	fclose(fp);

	// clean up memory:
	delete [ ] xcs;
	delete [ ] ycs;
	delete [ ] rs;
	delete [ ] hits;

	status = hipFree( dxcs );
		checkCudaErrors( status );
	status = hipFree( dycs );
		checkCudaErrors( status );
	status = hipFree( drs );
		checkCudaErrors( status );
	status = hipFree( dhits );
		checkCudaErrors( status );

	return 0;
}

